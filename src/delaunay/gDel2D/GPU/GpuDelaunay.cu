#include "hip/hip_runtime.h"
#include "../GpuDelaunay.h"

#include<iomanip>
#include<iostream>
#include <thrust/gather.h>

#include "KerCommon.h"
#include "KerDivision.h"
#include "KerPredicates.h"
#include "ThrustWrapper.h"

#include "../../Visualizer.h"

////
// GpuDel methods
////
void GpuDel::cleanup()
{
    thrust_free_all(); 

    _memPool.free(); 

    _pointVec.free();
    _constraintVec.free(); 
    _triVec.free(); 
    _oppVec.free(); 
    _triInfoVec.free(); 
    _orgPointIdx.free(); 
    _vertTriVec.free();
    _counters.free(); 
    _actConsVec.free(); 
        
    _orgFlipNum.clear(); 

    _dPredWrapper.cleanup(); 

    __circleCountVec.free(); 
    __rejFlipVec.free(); 

    _numActiveVec.clear(); 
    _numFlipVec.clear(); 
    _numCircleVec.clear(); 
    _timeCheckVec.clear(); 
    _timeFlipVec.clear(); 
}

void GpuDel::compute
(
Point2DVec &pointVec,
TriDVec*       output
)
{
    GDel2DInput input;
    input.noSort = true;
    input.noReorder = true;

    // Set L1 for kernels
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    _pointVec._ptr = pointVec._ptr;
    _pointVec._size = pointVec._size;
    _pointVec._capacity = pointVec._capacity;
    _pointVec._owned = false;

    _input  = &input;
    _output = new GDel2DOutput();
    //_output = output;

    initProfiling(); 

    startTiming( ProfDefault );

    initForFlip();
    splitAndFlip();
    outputToHost();



    stopTiming( ProfDefault, _output->stats.totalTime );

    if ( _input->isProfiling( ProfDetail ) )
    {
        std::cout << " FlipCompact time: ";
        _diagLogCompact.printTime();

        std::cout << std::endl;
        std::cout << " FlipCollect time: ";
        _diagLogCollect.printTime();

        std::cout << std::endl;
    }

    std::cout << "# GPU Triangles:     " << _triVec.size() << std::endl;

    *(output) = _triVec;
    _triVec._owned = false;

    cleanup();



    return;
}

void GpuDel::startTiming( ProfLevel level )
{
    if ( _input->isProfiling( level ) )
        _profTimer[ level ].start();
}

void GpuDel::pauseTiming( ProfLevel level )
{
    if ( _input->isProfiling( level ) )
        _profTimer[ level ].pause(); 
}

void GpuDel::stopTiming( ProfLevel level, double &accuTime )
{
    if ( _input->isProfiling( level ) )
    {
        _profTimer[ level ].stop(); 

        accuTime += _profTimer[ level ].value(); 
    }
}

void GpuDel::restartTiming( ProfLevel level, double &accuTime )
{
    stopTiming( level, accuTime ); 
    startTiming( level ); 
}

struct CompareX
{
	__device__ bool operator()( const Point2 &a, const Point2 &b ) const
	{
		return a._p[0] < b._p[0]; 
	}
};

struct Get2Ddist
{
	Point2		_a; 
	RealType	abx, aby; 

	Get2Ddist( const Point2 &a, const Point2 &b ) : _a(a)
	{
		abx = b._p[0] - a._p[0]; 
		aby = b._p[1] - a._p[1]; 
	}

	__device__ int operator()( const Point2 &c ) 
	{
		RealType acx = c._p[0] - _a._p[0]; 
		RealType acy = c._p[1] - _a._p[1]; 

		RealType dist = abx * acy - aby * acx; 

		return __float_as_int( fabs((float) dist) ); 
	}
};

RealType orient2dzero( const RealType *pa, const RealType *pb, const RealType *pc );

void GpuDel::constructInitialTriangles()
{
	// First, choose two extreme points along the X axis
	typedef Point2DVec::iterator Point2DIter; 

	thrust::pair< Point2DIter, Point2DIter > ret = thrust::minmax_element( 
        _pointVec.begin(), _pointVec.end(), CompareX() ); 

    int v0 = ret.first - _pointVec.begin(); 
	int v1 = ret.second - _pointVec.begin(); 

	const Point2 p0 = _pointVec[v0]; 
	const Point2 p1 = _pointVec[v1]; 

	// Find the furthest point from v0v1
	IntDVec distVec = _memPool.allocateAny<int>( _pointNum ); 

	distVec.resize( _pointVec.size() ); 

	thrust::transform( _pointVec.begin(), _pointVec.end(), distVec.begin(), Get2Ddist( p0, p1 ) ); 

	const int v2	= thrust::max_element( distVec.begin(), distVec.end() ) - distVec.begin(); 
	const Point2 p2 = _pointVec[v2]; 

    _memPool.release( distVec ); 

    if ( _input->isProfiling( ProfDebug ) )
	{
		std::cout << "Leftmost: " << v0 << " --> " << p0._p[0] << " " << p0._p[1] << std::endl; 
		std::cout << "Rightmost: " << v1 << " --> " << p1._p[0] << " " << p1._p[1] << std::endl; 
		std::cout << "Furthest 2D: " << v2 << " --> " << p2._p[0] << " " << p2._p[1] << std::endl; 
	}

	// Check to make sure the 4 points are not co-planar
	RealType ori = orient2dzero( p0._p, p1._p, p2._p ); 

	if ( ori == 0.0 )
	{
		std::cout << "Input too degenerate!!!\n" << std::endl; 
		exit(-1); 
	}

	if ( ortToOrient( ori ) == OrientNeg ) 
		std::swap( v0, v1 ); 

	// Compute the centroid of v0v1v2v3, to be used as the kernel point. 
	_ptInfty._p[0] = ( p0._p[0] + p1._p[0] + p2._p[0] ) / 3.0; 
	_ptInfty._p[1] = ( p0._p[1] + p1._p[1] + p2._p[1] ) / 3.0; 

    // Add the infinity point to the end of the list
    _infIdx = _pointNum - 1; 

    _pointVec.resize( _pointNum ); 
    _pointVec[ _infIdx ] = _ptInfty; 

	if ( _input->isProfiling( ProfDiag ) ) 
	{
		std::cout << "Kernel: " << _ptInfty._p[0] << " " << _ptInfty._p[1] << std::endl; 
	}

    // Initialize the predicate wrapper!!!
    _dPredWrapper.init( 
        toKernelPtr( _pointVec ), 
        _pointNum, 
        _input->noSort ? NULL : toKernelPtr( _orgPointIdx ), 
        _infIdx ); 

    setPredWrapperConstant( _dPredWrapper ); 

    // Create the initial triangulation
    Tri firstTri = { v0, v1, v2 }; 

    _triVec.expand( 4 );
    _oppVec.expand( 4 );
    _triInfoVec.expand( 4 );

    // Put the initial tets at the Inf list
    kerMakeFirstTri<<< 1, 1 >>>(
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
		firstTri, _infIdx
		);
    CudaCheckError();

    // Locate initial positions of points
    _vertTriVec.resize( _pointNum );

    IntDVec exactCheckVec = _memPool.allocateAny<int>( _pointNum ); 

    _counters.renew(); 

    kerInitPointLocationFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _vertTriVec ), 
        toKernelPtr( exactCheckVec ), 
        _counters.ptr(), 
        firstTri 
        ); 

    kerInitPointLocationExact<<< PredBlocksPerGrid, PredThreadsPerBlock >>>(
        toKernelPtr( _vertTriVec ), 
        toKernelPtr( exactCheckVec ), 
        _counters.ptr(), 
        firstTri
        ); 
    CudaCheckError();

    _memPool.release( exactCheckVec ); 

    _availPtNum = _pointNum - 4; 

    //Visualizer::instance()->addFrame( _pointVec, SegmentDVec(), _triVec, _infIdx );
}

void GpuDel::initForFlip()
{
    startTiming( ProfDefault ); 

    _pointNum = _pointVec.size();
    //_pointNum   = _input->pointVec.size() + 1;	// Plus the infinity point
    _triMax     = (int) ( _pointNum * 2 );

    // Copy points to GPU
    //_pointVec.resize( _pointNum );  // 1 additional slot for the infinity point
    //_pointVec.copyFromHost( _input->pointVec );

    // Copy constraints to GPU
    _constraintVec.copyFromHost( _input->constraintVec ); 

    // Allocate space
    _triVec.resize( _triMax );
    _oppVec.resize( _triMax );
    _triInfoVec.resize( _triMax );
    _counters.init( CounterNum ); 

    if ( _constraintVec.size() > 0 ) 
        _actConsVec.resize( _constraintVec.size() ); 

    if ( _input->isProfiling( ProfDiag ) )
    {
        __circleCountVec.resize( _triMax );
        __rejFlipVec.resize( _triMax );
    }

    // Preallocate some buffers in the pool
    _memPool.reserve<FlipItem>( _triMax );  // flipVec
    _memPool.reserve<int2>( _triMax );      // triMsgVec
    _memPool.reserve<int>( _pointNum );     // vertSphereVec
    _memPool.reserve<int>( _triMax );       // actTriVec
    _memPool.reserve<int>( _triMax );       // Two more for common use
    _memPool.reserve<int>( _triMax );       //

    if ( _constraintVec.size() > 0 ) 
        _memPool.reserve<int>( _triMax ); 

	// Find the min and max coordinate value
    typedef thrust::device_ptr< RealType > RealPtr; 
	RealPtr coords( ( RealType* ) toKernelPtr( _pointVec ) ); 
    thrust::pair< RealPtr, RealPtr> ret
        = thrust::minmax_element( coords, coords + _pointVec.size() * 2 ); 

    _minVal = *ret.first; 
    _maxVal = *ret.second; 

    if ( _input->isProfiling( ProfDebug ) ) 
    {
        std::cout << "_minVal = " << _minVal << ", _maxVal == " << _maxVal << std::endl; 
    }

    // Sort points along space curve
    if ( !_input->noSort )
    {
        stopTiming( ProfDefault, _output->stats.initTime ); 
        startTiming( ProfDefault ); 

        IntDVec valueVec = _memPool.allocateAny<int>( _pointNum );  
        valueVec.resize( _pointVec.size() );

        _orgPointIdx.resize( _pointNum ); 
        thrust::sequence( _orgPointIdx.begin(), _orgPointIdx.end(), 0 ); 

        thrust_transform_GetMortonNumber( 
            _pointVec.begin(), _pointVec.end(), valueVec.begin(),
            _minVal, _maxVal );

        thrust_sort_by_key( valueVec.begin(), valueVec.end(), 
            make_zip_iterator( make_tuple( 
                _orgPointIdx.begin(), _pointVec.begin() ) ) ); 

        _memPool.release( valueVec ); 

        stopTiming( ProfDefault, _output->stats.sortTime ); 
        startTiming( ProfDefault ); 
    }

    // Create first upper-lower triangles
	constructInitialTriangles(); 

    stopTiming( ProfDefault, _output->stats.initTime ); 

    return;
}

void GpuDel::doFlippingLoop( CheckDelaunayMode checkMode )
{
    startTiming( ProfDefault ); 

    _flipVec    = _memPool.allocateAny<FlipItem>( _triMax ); 
    _triMsgVec  = _memPool.allocateAny<int2>( _triMax ); 
    _actTriVec  = _memPool.allocateAny<int>( _triMax ); 

    _triMsgVec.assign( _triMax, make_int2( -1, -1 ) ); 

    int flipLoop = 0; 

    _actTriMode = ActTriMarkCompact;
    _diagLog    = &_diagLogCompact; 

    while ( doFlipping( checkMode ) ) 
        ++flipLoop; 

    stopTiming( ProfDefault, _output->stats.flipTime ); 

    relocateAll(); 

    _memPool.release( _triMsgVec ); 
    _memPool.release( _flipVec ); 
    _memPool.release( _actTriVec ); 
}

void GpuDel::initProfiling()
{
    _output->stats.reset(); 

    _diagLogCompact.reset(); 
    _diagLogCollect.reset(); 

    _numActiveVec.clear(); 
    _numFlipVec.clear(); 
    _timeCheckVec.clear(); 
    _timeFlipVec.clear(); 
}

void GpuDel::initForConstraintInsertion()
{
    if ( !_input->noSort )
    {
        // Update vertex indices of constraints
        IntDVec mapVec = _memPool.allocateAny<int>( _pointNum ); 
        mapVec.resize( _pointNum ); 

        thrust_scatterSequenceMap( _orgPointIdx, mapVec ); 

        thrust::device_ptr<int> segInt( (int *) toKernelPtr( _constraintVec ) ); 
        thrust::gather( segInt, segInt + _constraintVec.size() * 2, mapVec.begin(), segInt ); 

        _memPool.release( mapVec ); 

    //    // Sort the constraints
    //    const int constraintNum = _constraintVec.size(); 

    //    IntDVec keyVec = _memPool.allocateAny<int>( constraintNum ); 
    //    keyVec.resize( constraintNum ); 

    //    thrust::transform( _constraintVec.begin(), _constraintVec.end(), keyVec.begin(), GetConstraintMinVert() ); 

    //    thrust::sort_by_key( keyVec.begin(), keyVec.end(), _constraintVec.begin() ); 

    //    _memPool.release( keyVec ); 
    }

    // Construct 
    _vertTriVec.resize( _pointNum ); 

    kerMapTriToVert<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _triVec ), 
        toKernelPtr( _vertTriVec )
        ); 
    CudaCheckError(); 

    // Initialize list of active constraints
    thrust::sequence( _actConsVec.begin(), _actConsVec.end() ); 
}

bool GpuDel::markIntersections() 
{
    _counters.renew(); 

    kerMarkTriConsIntersectionFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actConsVec ), 
        toKernelPtr( _constraintVec ), 
        toKernelPtr( _triVec ), 
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( _vertTriVec ), 
        toKernelPtr( _triConsVec ), 
        _counters.ptr()
        ); 

    kerMarkTriConsIntersectionExact<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actConsVec ), 
        toKernelPtr( _constraintVec ), 
        toKernelPtr( _triVec ), 
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( _vertTriVec ), 
        toKernelPtr( _triConsVec ), 
        _counters.ptr()
        ); 
    CudaCheckError(); 

    return ( _counters[ CounterFlag ] == 1 ); 
}

void GpuDel::updatePairStatus()
{
    IntDVec exactVec = _memPool.allocateAny<int>( _triMax );  

    _counters.renew(); 

    kerUpdatePairStatusFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actTriVec ),
        toKernelPtr( _triConsVec ),
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( exactVec ), 
        _counters.ptr()
        );

    kerUpdatePairStatusExact<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actTriVec ),
        toKernelPtr( _triConsVec ),
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( exactVec ), 
        _counters.ptr()
        );
    CudaCheckError();

    _memPool.release( exactVec ); 
}

void GpuDel::checkConsFlipping( IntDVec& triVoteVec )
{
    IntDVec exactVec = _memPool.allocateAny<int>( _triMax );  

    _counters.renew(); 

    kerCheckConsFlippingFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actTriVec ), 
        toKernelPtr( _triConsVec ),
        toKernelPtr( _triInfoVec ), 
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ), 
        toKernelPtr( triVoteVec ),
        toKernelPtr( exactVec ), 
        _counters.ptr()
        );

    kerCheckConsFlippingExact<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actTriVec ), 
        toKernelPtr( _triConsVec ),
        toKernelPtr( _triInfoVec ), 
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ), 
        toKernelPtr( triVoteVec ),
        toKernelPtr( exactVec ), 
        _counters.ptr()
        );
    CudaCheckError();

    _memPool.release( exactVec ); 
}

bool GpuDel::doConsFlipping( int &flipNum )
{
    const int triNum  = _triVec.size();
    const int actNum  = _actTriVec.size(); 

    ///////
    // Vote for flips
    ///////
#pragma region Diagnostic
    if ( _input->isProfiling( ProfDiag ) )
        __rejFlipVec.assign( triNum, 0 );
#pragma endregion

    updatePairStatus(); 

    IntDVec triVoteVec = _memPool.allocateAny<int>( _triMax ); 
    triVoteVec.assign( triNum, INT_MAX );

    checkConsFlipping( triVoteVec ); 

    ////
    // Mark rejected flips
    ////
    IntDVec flipToTri = _memPool.allocateAny<int>( _triMax ); 

    flipToTri.resize( actNum );

    kerMarkRejectedConsFlips<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _actTriVec ),
        toKernelPtr( _triConsVec ), 
        toKernelPtr( triVoteVec ),
        toKernelPtr( _triInfoVec ), 
        toKernelPtr( _oppVec ),
        toKernelPtr( flipToTri ),
        _input->isProfiling( ProfDiag ) ? toKernelPtr( __rejFlipVec ) : NULL );
    CudaCheckError();

    _memPool.release( triVoteVec ); 

    ////
    // Compact flips
    ////
    IntDVec temp = _memPool.allocateAny<int>( _triMax, true ); 
    flipNum = compactIfNegative( flipToTri, temp ); 

    if ( 0 == flipNum ) 
    {
        _memPool.release( flipToTri ); 
        return false; 
    }

    ////
    // Expand flip vector
    ////
    int orgFlipNum = _flipVec.size(); 
    int expFlipNum = orgFlipNum + flipNum; 

    if ( expFlipNum > _flipVec.capacity() ) 
    {
        _flipVec.resize( 0 ); 
        _triMsgVec.assign( _triMax, make_int2( -1, -1 ) ); 

        orgFlipNum = 0; 
        expFlipNum = flipNum; 
    }

    _flipVec.grow( expFlipNum ); 

    // See doFlipping
    _triMsgVec.resize( _triVec.size() ); 

    ////
    // Flipping
    ////
#pragma region Diagnostic
    if ( _input->isProfiling( ProfDiag ) )
    {
        const int rejFlipNum = thrust_sum( __rejFlipVec );

        std::cout << "  ConsFlips: " << flipNum << " ( " << rejFlipNum << " )" 
            << std::endl;
    }
#pragma endregion

    // 32 ThreadsPerBlock is optimal
    kerFlip<<< BlocksPerGrid, 32 >>>( 
        toKernelArray( flipToTri ),
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ),
        NULL,
        toKernelPtr( _triMsgVec ),
        NULL,
        toKernelPtr( _flipVec ),
        toKernelPtr( _triConsVec ), 
        toKernelPtr( _vertTriVec ),
        orgFlipNum, 0
        ); 
    CudaCheckError(); 

    ////
    // Update oppTri
    ////
    kerUpdateOpp<<< BlocksPerGrid, 32 >>>(
        toKernelPtr( _flipVec ) + orgFlipNum,
        toKernelPtr( _oppVec ),
        toKernelPtr( _triMsgVec ),
        toKernelPtr( flipToTri ),
        orgFlipNum, flipNum
        ); 
    CudaCheckError();

    _memPool.release( flipToTri ); 

/////////////////////////////////////////////////////////////////////

    return true;   
}

void GpuDel::doInsertConstraints() 
{
    startTiming( ProfDefault ); 

    initForConstraintInsertion(); 

    const int triNum = _triVec.size(); 

    _triConsVec = _memPool.allocateAny<int>( triNum ); 
    _triConsVec.assign( triNum, -1 ); 

    _flipVec    = _memPool.allocateAny<FlipItem>( _triMax ); 
    _triMsgVec  = _memPool.allocateAny<int2>( _triMax ); 
    _actTriVec  = _memPool.allocateAny<int>( _triMax ); 

    _triMsgVec.assign( _triMax, make_int2( -1, -1 ) ); 

    int outerLoop  = 0; 
    int flipLoop   = 0; 
    int totFlipNum = 0; 
    int flipNum; 

    while ( markIntersections() ) 
    {
        if ( _input->isProfiling( ProfDiag ) )
            std::cout << "Iter " << ( outerLoop+1 ) << std::endl; 

        // VISUALIZATION
        if ( Visualizer::instance()->isEnable() ) 
        {
            pauseTiming( ProfNone ); 
            pauseTiming( ProfDefault ); 

            IntHVec triColorVec; 
            _triConsVec.copyToHost( triColorVec ); 

            for ( int i = 0; i < triColorVec.size(); ++i ) 
                if ( triColorVec[i] != -1 ) 
                    triColorVec[i] >>= 4; 

            Visualizer::instance()->addFrame( _pointVec, _constraintVec, _triVec, triColorVec, _infIdx ); 

            startTiming( ProfDefault ); 
            startTiming( ProfNone ); 
        }

        // Collect active triangles
        thrust_copyIf_IsNotNegative( _triConsVec, _actTriVec ); 

        int innerLoop = 0; 

        while ( doConsFlipping( flipNum ) )
        {
            totFlipNum += flipNum; 

            // VISUALIZATION
            if ( Visualizer::instance()->isEnable() ) 
            {
                pauseTiming( ProfNone ); 
                pauseTiming( ProfDefault ); 

                IntHVec triColorVec; 
                _triConsVec.copyToHost( triColorVec ); 

                for ( int i = 0; i < triColorVec.size(); ++i ) 
                    if ( triColorVec[i] != -1 ) 
                        triColorVec[i] >>= 4; 

                Visualizer::instance()->addFrame( _pointVec, _constraintVec, _triVec, triColorVec, _infIdx ); 

                startTiming( ProfDefault ); 
                startTiming( ProfNone ); 
            }

            ++flipLoop; 
            ++innerLoop; 

            if ( innerLoop == 5 ) break; 

            //if ( flipLoop == 1 ) break; 
        }

        ++outerLoop; 

        // Mark all the possibly modified triangles as Alive + Changed (3). 
        thrust_scatterConstantMap( _actTriVec, _triInfoVec, 3 ); 

        //if ( outerLoop == 5 ) break; 
    }

    //if ( outerLoop >= 20 )
    //{
    //    for ( int i = 0; i < _actTriVec.size(); ++i ) 
    //        std::cout << _actTriVec[i] << " "; 
    //    std::cout << std::endl; 
    //}

    if ( _input->isProfiling( ProfDiag ) )
        std::cout << "ConsFlip: Outer loop = " << outerLoop 
        << ", inner loop = " << flipLoop 
        << ", total flip = " << totFlipNum 
        << std::endl; 

    _memPool.release( _triConsVec ); 
    _memPool.release( _triMsgVec ); 
    _memPool.release( _actTriVec );        
    _memPool.release( _flipVec );        

    stopTiming( ProfDefault, _output->stats.constraintTime ); 
}

void GpuDel::splitAndFlip()
{
    int insLoop = 0;

    _doFlipping = !_input->insAll; 

    //////////////////
    while ( _availPtNum > 0 )
    //////////////////
    {
        ////////////////////////
        splitTri();
        ////////////////////////

        if ( _doFlipping ) 
            doFlippingLoop( CircleFastOrientFast ); 

        ++insLoop;
    }

    //////////////////////////////
    if ( !_doFlipping ) 
        doFlippingLoop( CircleFastOrientFast ); 

    markSpecialTris(); 
    doFlippingLoop( CircleExactOrientSoS ); 

    //////////////////////////////
    // Insert constraints if needed
    if ( _constraintVec.size() > 0 ) 
        doInsertConstraints(); 

    doFlippingLoop( CircleFastOrientFast ); 

    markSpecialTris(); 
    doFlippingLoop( CircleExactOrientSoS ); 

#pragma region Diagnostic
    if ( _input->isProfiling( ProfDiag ) )
    {
        std::cout << "\nInsert loops: " << insLoop << std::endl;

        std::cout << "Compact: " << std::endl; 
        _diagLogCompact.printCount(); 

        std::cout << "Collect: " << std::endl; 
        _diagLogCollect.printCount(); 
    }
#pragma endregion

    return;
}

void GpuDel::markSpecialTris()
{
    startTiming( ProfDetail ); 

    kerMarkSpecialTris<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _triInfoVec ), 
        toKernelPtr( _oppVec )
        );    
    CudaCheckError(); 

    stopTiming( ProfDetail, _diagLog->_t[ 0 ] ); 
}

void GpuDel::expandTri( int newTriNum )
{
    //*** Expand triangles
    _triVec.expand( newTriNum );
    _oppVec.expand( newTriNum );
    _triInfoVec.expand( newTriNum );
}

void GpuDel::splitTri()
{
    const int MaxSamplePerTri = 100; 

    startTiming( ProfDefault ); 

    ////
    // Rank points
    ////
    int triNum   = _triVec.size();
    int noSample = _pointNum; 
    
    if ( noSample / triNum > MaxSamplePerTri )
        noSample = triNum * MaxSamplePerTri; 

    IntDVec triCircleVec = _memPool.allocateAny<int>( _triMax ); 
    triCircleVec.assign( triNum, INT_MIN );

    IntDVec vertCircleVec = _memPool.allocateAny<int>( _pointNum ); 
    vertCircleVec.resize( noSample );

    kerVoteForPoint<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _vertTriVec ),
        toKernelPtr( _triVec ),
        toKernelPtr( vertCircleVec ),
        toKernelPtr( triCircleVec ),
        noSample
        );
    CudaCheckError();

    IntDVec triToVert = _memPool.allocateAny<int>( _triMax ); 
    triToVert.assign( triNum, INT_MAX );

    kerPickWinnerPoint<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _vertTriVec ),
        toKernelPtr( vertCircleVec ),
        toKernelPtr( triCircleVec ),
        toKernelPtr( triToVert ),
        noSample
        );
    CudaCheckError();

    _memPool.release( vertCircleVec ); 
    _memPool.release( triCircleVec ); 

    ////
    // Collect triangles with insertions
    ////
    IntDVec splitTriVec = _memPool.allocateAny<int>( _pointNum ); 
    _insNum = thrust_copyIf_TriHasVert( triToVert, splitTriVec ); 

    const int extraTriNum   = DIM * _insNum;
    const int splitTriNum   = triNum + extraTriNum;

    if ( _input->isProfiling( ProfDiag ) )
    {
        std::cout << "Insert: " << _insNum 
        << " Tri from: " << triNum
        << " to: " << splitTriNum << std::endl;
    }

    // If there's just a few points
    if ( _availPtNum - _insNum < _insNum && 
        _insNum < 0.1 * _pointNum ) 
    {
        _doFlipping = false; 
        //std::cout << "Stop flipping!" << std::endl; 
    }

    if ( !_input->noReorder && _doFlipping ) 
    {
        stopTiming( ProfDefault, _output->stats.splitTime ); 

        shiftTri( triToVert, splitTriVec ); 

        triNum = -1;    // Mark that we have shifted the array

        startTiming( ProfDefault ); 
    }

    ////
    // Make map
    ////
    IntDVec insTriMap = _memPool.allocateAny<int>( _triMax ); 

    insTriMap.assign( ( triNum < 0 ) ? splitTriNum : triNum, -1 );

    thrust_scatterSequenceMap( splitTriVec, insTriMap ); 

    ////
    // Expand if space needed
    ////
    expandTri( splitTriNum );

    ////
    // Update the location of the points
    ////
    stopTiming( ProfDefault, _output->stats.splitTime ); 
    startTiming( ProfDefault ); 

    IntDVec exactCheckVec = _memPool.allocateAny<int>( _pointNum ); 

    _counters.renew(); 

    kerSplitPointsFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _vertTriVec ),
        toKernelPtr( triToVert ),
        toKernelPtr( _triVec ),
        toKernelPtr( insTriMap ),
        toKernelPtr( exactCheckVec ), 
        _counters.ptr(),
        triNum, _insNum
        );

    kerSplitPointsExactSoS<<< PredBlocksPerGrid, PredThreadsPerBlock >>>(
        toKernelPtr( _vertTriVec ),
        toKernelPtr( triToVert ),
        toKernelPtr( _triVec ),
        toKernelPtr( insTriMap ),
        toKernelPtr( exactCheckVec ), 
        _counters.ptr(),
        triNum, _insNum
        );
    CudaCheckError();

    _memPool.release( exactCheckVec ); 

    stopTiming( ProfDefault, _output->stats.relocateTime ); 
    startTiming( ProfDefault ); 

    ////
    // Split old into new triangle and copy them to new array
    ////
    kerSplitTri<<< BlocksPerGrid, 32 >>>(
        toKernelArray( splitTriVec ),
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( insTriMap ),
        toKernelPtr( triToVert ),
        triNum, _insNum
        );
    CudaCheckError();

    _memPool.release( triToVert ); 
    _memPool.release( insTriMap );
    _memPool.release( splitTriVec ); 

    _availPtNum -= _insNum; 

    stopTiming( ProfDefault, _output->stats.splitTime ); 

    Visualizer::instance()->addFrame( _pointVec, SegmentDVec(), _triVec, _infIdx ); 

    return;
}

bool GpuDel::doFlipping( CheckDelaunayMode checkMode )
{
    startTiming( ProfDetail ); 

    ++_diagLog->_flipLoop; 

    const int triNum  = _triVec.size();

    ////
    // Compact active triangles
    ////

    switch ( _actTriMode ) 
    {
    case ActTriMarkCompact: 
        thrust_copyIf_IsActiveTri( _triInfoVec, _actTriVec ); 
        break; 

    case ActTriCollectCompact: 
        IntDVec temp = _memPool.allocateAny<int>( _triMax, true ); 
        compactIfNegative( _actTriVec, temp );
        break; 
    }

    int orgActNum = _actTriVec.size(); 

#pragma region Diagnostic
    if ( _input->isProfiling( ProfDiag ) )
    {
        _numActiveVec.push_back( orgActNum ); 

        if ( orgActNum == 0 || ( checkMode != CircleExactOrientSoS && 
            orgActNum < PredBlocksPerGrid * PredThreadsPerBlock ) ) 
        {
            _numFlipVec.push_back( 0 ); 
            _timeCheckVec.push_back( 0.0 ); 
            _timeFlipVec.push_back( 0.0 ); 
            _numCircleVec.push_back( 0 ); 
        }
    }
#pragma endregion

    restartTiming( ProfDetail, _diagLog->_t[ 0 ] ); 
/////////////////////////////////////////////////////////////////////
    ////
    // Check actNum, switch mode or quit if necessary
    ////

    // No more work
    if ( 0 == orgActNum )
        return false;

    // Little work, leave it for the Exact iterations
    if ( checkMode != CircleExactOrientSoS && 
        orgActNum < PredBlocksPerGrid * PredThreadsPerBlock ) 
        return false; 

    // See if there's little work enough to switch to collect mode. 
    // Safety check: make sure there's enough space to collect
    if ( orgActNum < BlocksPerGrid * ThreadsPerBlock &&
        orgActNum * 2 < _actTriVec.capacity() &&
        orgActNum * 2 < triNum )
    {
        _actTriMode = ActTriCollectCompact; 
        _diagLog    = &_diagLogCollect; 
    }
    else
    {
        _actTriMode = ActTriMarkCompact; 
        _diagLog    = &_diagLogCompact; 
    }

    ////
    // Vote for flips
    ////

#pragma region Diagnostic
    if ( _input->isProfiling( ProfDiag ) )
    {
        __circleCountVec.assign( triNum, 0 );
        __rejFlipVec.assign( triNum, 0 );
    }
#pragma endregion

    IntDVec triVoteVec = _memPool.allocateAny<int>( _triMax ); 
    triVoteVec.assign( triNum, INT_MAX );

    dispatchCheckDelaunay( checkMode, orgActNum, triVoteVec ); 

    double prevTime = _diagLog->_t[ 1 ]; 

    restartTiming( ProfDetail, _diagLog->_t[ 1 ] );
/////////////////////////////////////////////////////////////////////
    ////
    // Mark rejected flips
    ////

    IntDVec flipToTri = _memPool.allocateAny<int>( _triMax ); 

    flipToTri.resize( orgActNum );
    
    kerMarkRejectedFlips<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelPtr( _actTriVec ),
        toKernelPtr( _oppVec ),
        toKernelPtr( triVoteVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( flipToTri ),
        orgActNum,
        _input->isProfiling( ProfDiag ) ? toKernelPtr( __rejFlipVec ) : NULL );
    CudaCheckError();

    _memPool.release( triVoteVec ); 
    
    restartTiming( ProfDetail, _diagLog->_t[ 2 ] ); 
/////////////////////////////////////////////////////////////////////
    ////
    // Compact flips
    ////
    IntDVec temp = _memPool.allocateAny<int>( _triMax, true ); 
    const int flipNum = compactIfNegative( flipToTri, temp ); 

    if ( _input->isProfiling( ProfDiag ) )
    {
        _numFlipVec.push_back( flipNum ); 
        _timeCheckVec.push_back( _diagLog->_t[ 1 ] - prevTime ); 
    }

    restartTiming( ProfDetail, _diagLog->_t[ 3 ] );  
/////////////////////////////////////////////////////////////////////
    ////
    // Preparation for the actual flipping. Include several steps
    ////

#pragma region Diagnostic
    if ( _input->isProfiling( ProfDiag ) )
    {
        const int circleNum = thrust_sum( __circleCountVec );
        _diagLog->_circleCount += circleNum; 
        const int rejFlipNum = thrust_sum( __rejFlipVec );
        _diagLog->_rejFlipCount += rejFlipNum;

        _diagLog->_totFlipNum   += flipNum;

        std::cout << "Acts: " << orgActNum
			<< " Flips: " << flipNum << " ( " << rejFlipNum << " )" 
            << " circle: " << circleNum 
            << " Exact: " 
            << ( checkMode == CircleExactOrientSoS ? _counters[ CounterExact ] : -1 )
            << std::endl;

        _numCircleVec.push_back( circleNum ); 

        startTiming( ProfDetail ); 
    }
#pragma endregion

    if ( 0 == flipNum )
    {
        _numCircleVec.push_back( 0 ); 
        _timeFlipVec.push_back( 0 ); 
        _memPool.release( flipToTri ); 
        return false;
    }

    // Expand flip vector
    int orgFlipNum = _flipVec.size(); 
    int expFlipNum = orgFlipNum + flipNum; 

    if ( expFlipNum > _flipVec.capacity() ) 
    {
        stopTiming( ProfDetail, _diagLog->_t[ 4 ] ); 
        stopTiming( ProfDefault, _output->stats.flipTime ); 
           relocateAll(); 
        startTiming( ProfDefault ); 
        startTiming( ProfDetail ); 

        orgFlipNum = 0; 
        expFlipNum = flipNum; 
    }

    _flipVec.grow( expFlipNum ); 

    // _triMsgVec contains two components. 
    // - .x is the encoded new neighbor information
    // - .y is the flipIdx as in the flipVec (i.e. globIdx)
    // As such, we do not need to initialize it to -1 to 
    // know which tris are not flipped in the current rount. 
    // We can rely on the flipIdx being > or < than orgFlipIdx. 
    // Note that we have to initialize everything to -1 
    // when we clear the flipVec and reset the flip indexing. 
    //
    _triMsgVec.resize( _triVec.size() ); 

    ////
    // Expand active tri vector
    ////
    if ( _actTriMode == ActTriCollectCompact ) 
        _actTriVec.grow( orgActNum + flipNum );

    restartTiming( ProfDetail, _diagLog->_t[ 4 ] ); 
/////////////////////////////////////////////////////////////////////
    ////
    // Flipping
    ////

    // 32 ThreadsPerBlock is optimal
    kerFlip<<< BlocksPerGrid, 32 >>>( 
        toKernelArray( flipToTri ),
        toKernelPtr( _triVec ),
        toKernelPtr( _oppVec ),
        toKernelPtr( _triInfoVec ),
        toKernelPtr( _triMsgVec ),
        ( _actTriMode == ActTriCollectCompact ) ? toKernelPtr( _actTriVec ) : NULL,
        toKernelPtr( _flipVec ),
        NULL, NULL,
        orgFlipNum, orgActNum
        ); 
    CudaCheckError(); 

    _orgFlipNum.push_back( orgFlipNum ); 

    ////
    // Update oppTri
    ////

    kerUpdateOpp<<< BlocksPerGrid, 32 >>>(
        toKernelPtr( _flipVec ) + orgFlipNum,
        toKernelPtr( _oppVec ),
        toKernelPtr( _triMsgVec ),
        toKernelPtr( flipToTri ),
        orgFlipNum, flipNum
        ); 
    CudaCheckError();

    _memPool.release( flipToTri ); 

    prevTime = _diagLog->_t[ 5 ]; 

    stopTiming( ProfDetail, _diagLog->_t[ 5 ] ); 

    if ( _input->isProfiling( ProfDiag ) )
        _timeFlipVec.push_back( _diagLog->_t[ 5 ] - prevTime ); 
/////////////////////////////////////////////////////////////////////

    Visualizer::instance()->addFrame( _pointVec, SegmentDVec(), _triVec, _infIdx ); 

    return true;
}

void GpuDel::dispatchCheckDelaunay
( 
CheckDelaunayMode   checkMode, 
int                 orgActNum,
IntDVec&            triVoteVec
) 
{
    switch ( checkMode ) 
    {
    case CircleFastOrientFast: 
        kerCheckDelaunayFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
            toKernelPtr( _actTriVec ),
            toKernelPtr( _triVec ),
            toKernelPtr( _oppVec ),
            toKernelPtr( _triInfoVec ),
            toKernelPtr( triVoteVec ),
            orgActNum,
            _input->isProfiling( ProfDiag ) ? toKernelPtr( __circleCountVec ) : NULL
            );
        CudaCheckError();
        break; 

    case CircleExactOrientSoS:
        // Reuse this array to save memory
        Int2DVec &exactCheckVi = _triMsgVec; 

        _counters.renew(); 

        kerCheckDelaunayExact_Fast<<< BlocksPerGrid, ThreadsPerBlock >>>(
            toKernelPtr( _actTriVec ),
            toKernelPtr( _triVec ),
            toKernelPtr( _oppVec ),
            toKernelPtr( _triInfoVec ),
            toKernelPtr( triVoteVec ),
            toKernelPtr( exactCheckVi ), 
            orgActNum,
            _counters.ptr(), 
            _input->isProfiling( ProfDiag ) ? toKernelPtr( __circleCountVec ) : NULL
            );

        kerCheckDelaunayExact_Exact<<< PredBlocksPerGrid, PredThreadsPerBlock >>>(
            toKernelPtr( _triVec ),
            toKernelPtr( _oppVec ),
            toKernelPtr( triVoteVec ),
            toKernelPtr( exactCheckVi ), 
            _counters.ptr(), 
            _input->isProfiling( ProfDiag ) ? toKernelPtr( __circleCountVec ) : NULL
            );
        CudaCheckError();

        break; 
    }
}

template< typename T >
__global__ void 
kerShift
(
KerIntArray shiftVec, 
T*          src, 
T*          dest
) 
{
    for ( int idx = getGlobThreadIdx(); idx < shiftVec._num; idx += getThreadNum() )
    {
        const int shift = shiftVec._arr[ idx ]; 

        dest[ idx + shift ] = src[ idx ]; 
    }
}

template< typename T > 
void GpuDel::shiftExpandVec( IntDVec &shiftVec, DevVector< T > &dataVec, int size )
{
    DevVector< T > tempVec = _memPool.allocateAny<T>( size ); 

    tempVec.resize( size );

    kerShift<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( shiftVec ), 
        toKernelPtr( dataVec ),
        toKernelPtr( tempVec )
        ); 
    CudaCheckError(); 

    dataVec.copyFrom( tempVec ); 

    _memPool.release( tempVec ); 
}

void GpuDel::shiftOppVec( IntDVec &shiftVec, TriOppDVec &dataVec, int size )
{
    TriOppDVec tempVec = _memPool.allocateAny< TriOpp >( size ); 

    tempVec.resize( size );

    kerShiftOpp<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( shiftVec ), 
        toKernelPtr( dataVec ),
        toKernelPtr( tempVec ),
        size
        ); 
    CudaCheckError(); 

    dataVec.copyFrom( tempVec ); 

    _memPool.release( tempVec ); 
}

void GpuDel::shiftTri( IntDVec &triToVert, IntDVec &splitTriVec )
{
    startTiming( ProfDefault ); 

    const int triNum = _triVec.size() + 2 * splitTriVec.size(); 

    IntDVec shiftVec = _memPool.allocateAny<int>( _triMax ); 

    thrust_scan_TriHasVert( triToVert, shiftVec ); 

    shiftExpandVec( shiftVec, _triVec, triNum ); 
    shiftExpandVec( shiftVec, _triInfoVec, triNum ); 
    shiftExpandVec( shiftVec, triToVert, triNum ); 
    shiftOppVec( shiftVec, _oppVec, triNum ); 

    kerShiftTriIdx<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _vertTriVec ),
        toKernelPtr( shiftVec )
        ); 
    CudaCheckError(); 

    kerShiftTriIdx<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( splitTriVec ),
        toKernelPtr( shiftVec )
        ); 
    CudaCheckError(); 

    _memPool.release( shiftVec ); 

    stopTiming( ProfDefault, _output->stats.sortTime ); 
}

void GpuDel::relocateAll()
{
    if ( _flipVec.size() == 0 ) 
        return ; 

    startTiming( ProfDefault ); 

    if ( _availPtNum > 0 ) 
    {
        const int triNum = _triVec.size(); 

        IntDVec triToFlip = _memPool.allocateAny<int>( _triMax ); 
        triToFlip.assign( triNum, -1 ); 

        // Rebuild the pointers from back to forth
        int nextFlipNum = _flipVec.size(); 

        for ( int i = _orgFlipNum.size() - 1; i >= 0; --i ) 
        {
            int prevFlipNum = _orgFlipNum[ i ]; 
            int flipNum     = nextFlipNum - prevFlipNum; 

            kerUpdateFlipTrace<<< BlocksPerGrid, ThreadsPerBlock >>>(
                toKernelPtr( _flipVec ), 
                toKernelPtr( triToFlip ),
                prevFlipNum, 
                flipNum 
                ); 

            nextFlipNum = prevFlipNum; 
        }
        CudaCheckError(); 
        
        // Relocate points
        IntDVec exactCheckVec = _memPool.allocateAny<int>( _pointNum ); 

        _counters.renew(); 

        kerRelocatePointsFast<<< BlocksPerGrid, ThreadsPerBlock >>>(
            toKernelArray( _vertTriVec ),
            toKernelPtr( triToFlip ),
            toKernelPtr( _flipVec ),
            toKernelPtr( exactCheckVec ), 
            _counters.ptr()
            );

        kerRelocatePointsExact<<< BlocksPerGrid, ThreadsPerBlock >>>(
            toKernelPtr( _vertTriVec ),
            toKernelPtr( triToFlip ),
            toKernelPtr( _flipVec ),
            toKernelPtr( exactCheckVec ), 
            _counters.ptr()
            );
        CudaCheckError();
    
        _memPool.release( exactCheckVec ); 
        _memPool.release( triToFlip ); 
    }

    // Just clean up the flips
    _flipVec.resize( 0 ); 
    _orgFlipNum.clear(); 

    // Reset the triMsgVec
    _triMsgVec.assign( _triMax, make_int2( -1, -1 ) ); 

    stopTiming( ProfDefault, _output->stats.relocateTime ); 
}

void GpuDel::compactTris()
{
    const int triNum = _triVec.size(); 

    IntDVec prefixVec = _memPool.allocateAny<int>( _triMax ); 
    
    prefixVec.resize( triNum ); 

    thrust_scan_TriAliveStencil( _triInfoVec, prefixVec ); 

    int newTriNum = prefixVec[ triNum - 1 ];
    int freeNum   = triNum - newTriNum; 

    IntDVec freeVec = _memPool.allocateAny<int>( _triMax ); 

    freeVec.resize( freeNum ); 

    kerCollectFreeSlots<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelPtr( _triInfoVec ), 
        toKernelPtr( prefixVec ),
        toKernelPtr( freeVec ),
        newTriNum
        ); 
    CudaCheckError(); 

    // Make map
    kerMakeCompactMap<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _triInfoVec ), 
        toKernelPtr( prefixVec ),
        toKernelPtr( freeVec ),
        newTriNum
        ); 
    CudaCheckError(); 

    // Reorder the tets
    kerCompactTris<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _triInfoVec ), 
        toKernelPtr( prefixVec ), 
        toKernelPtr( _triVec ), 
        toKernelPtr( _oppVec ),
        newTriNum
        ); 
    CudaCheckError(); 

    _triInfoVec.resize( newTriNum ); 
    _triVec.resize( newTriNum ); 
    _oppVec.resize( newTriNum ); 

    _memPool.release( freeVec ); 
    _memPool.release( prefixVec ); 
}

void GpuDel::outputToHost()
{
    startTiming( ProfDefault ); 

    kerMarkInfinityTri<<< BlocksPerGrid, ThreadsPerBlock >>>(
        toKernelArray( _triVec ), 
        toKernelPtr( _triInfoVec ),
        toKernelPtr( _oppVec ),
        _infIdx
        ); 
    CudaCheckError(); 

    compactTris(); 

    if ( !_input->noSort ) 
    {
        // Change the indices back to the original order
        kerUpdateVertIdx<<< BlocksPerGrid, ThreadsPerBlock >>>(
            toKernelArray( _triVec ), 
            toKernelPtr( _triInfoVec ),
            toKernelPtr( _orgPointIdx )
            ); 
        CudaCheckError(); 
    }

    ////
    // Copy to host  
    //_triVec.copyToHost( _output->triVec );
    //_oppVec.copyToHost( _output->triOppVec );

    // Output Infty point
    _output->ptInfty = _ptInfty; 

    stopTiming( ProfDefault, _output->stats.outTime ); 
    
    ////

    return;
}
