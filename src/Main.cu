#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <stdint.h>

#include "Elas.h"
#include "Image.h"
#include "Utils.h"

using namespace cv;
using namespace std;

void showDisparityMap(float* data, int width, int height, const char* file){
    // find maximum disparity for scaling output disparity images to [0..255]
    float disp_max = 0;
    for (int32_t i=0; i<width*height; i++) {
      if (data[i]>disp_max) disp_max = data[i];
    }

    // copy float to uchar
    uchar *_data = (uchar*) malloc(width*height*sizeof(uchar));
    for (int32_t i=0; i<width*height; i++) {
      _data[i] = (uint8_t)max(255.0*data[i]/disp_max,0.0);
    }


    Mat img(height, width, CV_8UC1, _data);
    applyColorMap(img, img, COLORMAP_JET);
    namedWindow( file , WINDOW_AUTOSIZE );
    imshow( file , img );
    waitKey(0);
    destroyWindow(file);
}

void printDeviceProp(){
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Async Engines: %d\n",
               prop.asyncEngineCount);
        printf("  Concurrent Kernels: %d\n",
               prop.concurrentKernels);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Block Size (X, Y, Z): (%d, %d, %d)\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max Grid Size (X, Y, Z): (%d, %d, %d)\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("  Shared Mem Per Block: %d bytes\n", prop.sharedMemPerBlock);
        printf("  Registers Per Block: %d\n\n", prop.regsPerBlock);
    }
    hipFree(0);
}

void process (const char* file_1,const char* file_2) {
    cout << "Processing: " << file_1 << ", " << file_2 << endl;

    // load images
    image<uint8_t> *I1,*I2;
    I1 = loadPGM(file_1);
    I2 = loadPGM(file_2);

    // check for correct size
    if (I1->width()<=0 || I1->height() <=0 || I2->width()<=0 || I2->height() <=0 ||
        I1->width()!=I2->width() || I1->height()!=I2->height()) {
      cout << "ERROR: Images must be of same size, but" << endl;
      cout << "       I1: " << I1->width() <<  " x " << I1->height() <<
                   ", I2: " << I2->width() <<  " x " << I2->height() << endl;
      delete I1;
      delete I2;
      return;
    }

    // get image width and height
    int32_t width  = I1->width();
    int32_t height = I1->height();

    // allocate memory for disparity images
    const int32_t dims[2] = {width,height};
    float* D1_data = (float*)malloc(width*height*sizeof(float));
    float* D2_data = (float*)malloc(width*height*sizeof(float));

    Elas elas;
    elas.process(I1->data, I2->data, D1_data, D2_data, dims);

    showDisparityMap(D1_data, width, height, file_1);
}



int main( int argc, char* argv[] )
{

    printDeviceProp();

    // run demo
    if (argc==2 && !strcmp(argv[1],"demo")) {
      process("img/teddy_left.pgm",   "img/teddy_right.pgm");
      process("img/cones_left.pgm",   "img/cones_right.pgm");
      process("img/aloe_left.pgm",    "img/aloe_right.pgm");
      process("img/raindeer_left.pgm","img/raindeer_right.pgm");
      cout << "... done!" << endl;

    // compute disparity from input pair
    } else if (argc==3) {
      process(argv[1],argv[2]);
      cout << "... done!" << endl;

    // display help
    } else {
      cout << endl;
      cout << "ELAS demo program usage: " << endl;
      cout << "./elas demo ................ process all test images (image dir)" << endl;
      cout << "./elas left.pgm right.pgm .. process a single stereo pair" << endl;
      cout << "./elas -h .................. shows this help" << endl;
      cout << endl;
      cout << "Note: All images must be pgm greylevel images. All output" << endl;
      cout << "      disparities will be scaled such that disp_max = 255." << endl;
      cout << endl;
    }

    return 0;
}
